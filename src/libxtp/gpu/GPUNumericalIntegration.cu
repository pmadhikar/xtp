#include <votca/xtp/gpu/GPUNumericalIntegration.cuh>
#include <votca/xtp/aoshell.h>
#include <cstdlib>

namespace votca { namespace xtp { namespace gpu {

// GPUNumericalIntegration
GPUNumericalIntegration::GPUNumericalIntegration(const AOBasis& aob):
_shellMap(aob.getNumofShells(), -1)
{
    // first check if there is a gpu and select it
    
    CUDA_API_CALL(hipSetDevice(0), true); // add some code here later to inteligently
                                           // select a gpu
    _device = 0; 
    // Later we need to also dissallow this gpu from begin used
    // by other processes...
    unsigned int numFuncs = 0;
    unsigned int shellInd = 0;
    for (AOBasis::AOShellIterator row = aob.firstShell(); row != aob.lastShell(); row++){
        int numFunInShell = 0;
        const AOShell* shell = aob.getShell(row); 
        const std::string shell_type = shell->getType();
        for (const char& c : shell_type){
            switch (c){
            case 'S':
                numFunInShell += 1;
                _gpuAOB.nS += 1; 
            case 'P':
                numFunInShell += 3;
                _gpuAOB.nP += 1; 
            case 'D':
                numFunInShell += 5;
                _gpuAOB.nD += 1; 
            case 'F':
                numFunInShell += 7;
                _gpuAOB.nF += 1; 
            case 'G':
                numFunInShell += 9;
                _gpuAOB.nG += 1; 
            case 'H':
                std::cerr << "H functions not implemented at the moment!" << std::endl;
                exit(EXIT_FAILURE);
            default:
                std::cerr << "Single shell type " << c << "unknown" << std::endl;
                exit(EXIT_FAILURE);
            }
        }
        numFuncs += numFunInShell;
        _shellMap[shellInd] = shell->getStartIndex();

        allocateGPUAOBasis(_gpuAOB);
    }
    
}

GPUNumericalIntegration::~GPUNumericalIntegration(){
    freeGPUAOBasis(_gpuAOB);
}

}}}
