#include <algorithm>
#include <string>
#include <votca/xtp/aoshell.h>
#include <thrust/host_vector.h>
#include <votca/xtp/gpu/GPUNumericalIntegration.cuh>

namespace votca { namespace xtp { namespace gpu {

GPUNumericalIntegration::GPUNumericalIntegration(const AOBasis& aob): _device(0), _aob(&aob){
    CUDA_API_CALL(hipSetDevice(_device), true); // add some code here later to intelligently
                                                 // select a gpu
    
    // Later we need to also dissallow this gpu from begin used
    // by other processes (optionally?)...
    
}

}}}
